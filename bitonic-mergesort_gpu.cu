/**
 * Sorts a list of numbers using the bitonic merge sort algorithm on the GPU.
 * 
 * Code mostly from https://gist.github.com/mre/1392067 with some modifications.
 * 
 * Compile with: nvcc -arch=sm_86 -O3 utilities.c bitonic-mergesort_gpu.cu -o bitonic-mergesort_gpu
 * Run with:
*/

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <stdbool.h>
#include <time.h>
#include <math.h>

// #include "utilities.h"

#include <hip/hip_runtime.h>

#define BLOCK_SIZE 512

double* create_array(int n) {
    double *arr = (double *)malloc(n * sizeof(double));
    if (arr == NULL) {
        printf("Error: Unable to allocate memory\n");
        return NULL;
    }

    srand(time(NULL));
    for (int i = 0; i < n; i++) {
        arr[i] = rand() / (double)RAND_MAX;
    }

    return arr;
}

bool is_sorted(double *arr, int n) {
    for (int i = 0; i < n - 1; i++) {
        if (arr[i] > arr[i + 1]) {
            return false;
        }
    }

    return true;
}

int lowest_power_of_two(int n) {
    int m = 1;
    while (m < n) {
        m *= 2;
    }
    return m;
}

void extend_array(double *arr, int n, int m) {

    for (int i = n; i < m; i++) {
        arr[i] = INFINITY;
    }
}

__global__ void bitonic_sort_step(double *dev_values, int j, int k) {
    unsigned int i, ixj; /* Sorting partners: i and ixj */
    i = threadIdx.x + blockDim.x * blockIdx.x;
    ixj = i^j;

    /* The threads with the lowest ids sort the array. */
    if ((ixj)>i) {
        if ((i&k)==0) {
            /* Sort ascending */
            if (dev_values[i]>dev_values[ixj]) {
                /* exchange(i,ixj); */
                double temp = dev_values[i];
                dev_values[i] = dev_values[ixj];
                dev_values[ixj] = temp;
            }
        }
        if ((i&k)!=0) {
            /* Sort descending */
            if (dev_values[i]<dev_values[ixj]) {
                /* exchange(i,ixj); */
                double temp = dev_values[i];
                dev_values[i] = dev_values[ixj];
                dev_values[ixj] = temp;
            }
        }
    }
}

void bitonic_sort(double *values, int n) {
    double *dev_values;
    size_t size = n * sizeof(double);

    hipMalloc((void**) &dev_values, size);
    hipMemcpy(dev_values, values, size, hipMemcpyHostToDevice);

    // int block = min(BLOCK_SIZE, lowest_power_of_two(n) / 2);

    dim3 blocks(BLOCK_SIZE, 1);
    dim3 threads(n / BLOCK_SIZE, 1);

    int j, k;
    /* Major step */
    for (k = 2; k <= n; k <<= 1) {
        /* Minor step */
        for (j=k>>1; j>0; j=j>>1) {
            bitonic_sort_step<<<blocks, threads>>>(dev_values, j, k);
        }
    }

    hipMemcpy(values, dev_values, size, hipMemcpyDeviceToHost);
    hipFree(dev_values);

    hipDeviceReset();
}

int main(int argc, char *argv[]) {
    if (argc != 2) {
        printf("Usage: %s array-length\n", argv[0]);
        return 1;
    }

    struct timespec start, end;

    int n = atoi(argv[1]);
    double *arr = create_array(n);
    if (arr == NULL) {
        printf("Error: Unable to allocate memory\n");
        return 1;
    }

    // print the array
    // for (int i = 0; i < n; i++) {
    //     printf("%f ", arr[i]);
    // }
    // printf("\n");

    clock_gettime(CLOCK_MONOTONIC, &start);

    // make the array length power of two by filling with INFINITY
    int m = lowest_power_of_two(n);
    arr = (double *)realloc(arr, m * sizeof(double));
    extend_array(arr, n, m);

    bitonic_sort(arr, m);

    clock_gettime(CLOCK_MONOTONIC, &end);

    // for (int i = 0; i < n; i++) {
    //     printf("%f ", arr[i]);
    // }
    // printf("\n");

    if (!is_sorted(arr, n)) {
        printf("Error: Array is not sorted\n");
        return 1;
    }

    double time_taken = end.tv_sec-start.tv_sec+(end.tv_nsec-start.tv_nsec)/1000000000.0;

    printf("Time taken: %f\n", time_taken);

    free(arr);

    return 0;


}